
#include <hip/hip_runtime.h>
#include<iostream>
using namespace std;

#define THREADS_PER_BLOCK 256

__global__ void mean_per_block(int *a_d,int *b_d,int n){
    int block = blockDim.x*blockIdx.x;
    int mean=0,sum=0;
    for(int i=block;i<min(block+blockDim.x,n);i++){
        
            sum+=a_d[i];
        
    }
    b_d[blockIdx.x]=sum/blockDim.x;
}

int main() {
    int n;
    cout<<"Enter the no of elements";
    cin>>n;

    int *arr = new int[n];
    for(int i=0;i<n;i++){
        arr[i]=i+1;
    }
    
    int no_of_blocks = (n+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK;
    int size = n*sizeof(int);
    int *arr_d,*b_d;
    hipMalloc(&arr_d,size);
    hipMalloc(&b_d,no_of_blocks*sizeof(int));
    hipMemcpy(arr_d,arr,size,hipMemcpyHostToDevice);

    while(n>1){
        mean_per_block<<<no_of_blocks,THREADS_PER_BLOCK>>>(arr_d,b_d,n);
        n=(n+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK;
        hipMemcpy(arr_d,b_d,no_of_blocks*sizeof(int),hipMemcpyDeviceToDevice);
    }
    int ans;
    hipMemcpy(&ans,arr_d,sizeof(int),hipMemcpyDeviceToHost);
    cout<<ans;
}