
#include <hip/hip_runtime.h>
#include<iostream>
using namespace std;

#define THREADS_PER_BLOCK 256

__global__ void max_per_block(int *a_d,int *b_d,int n){
    int block = blockDim.x*blockIdx.x;
    int max=0;
    for(int i=block;i<min(block+blockDim.x,n);i++){
        if(max<a_d[i]){
            max=a_d[i];
        }
    }
    b_d[blockIdx.x]=max;
}

int main() {
    int n;
    cout<<"Enter the no of elements";
    cin>>n;

    int *arr = new int[n];
    for(int i=0;i<n;i++){
        arr[i]=i+1;
    }
    
    int no_of_blocks = (n+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK;
    int size = n*sizeof(int);
    int *arr_d,*b_d;
    hipMalloc(&arr_d,size);
    hipMalloc(&b_d,no_of_blocks*sizeof(int));
    hipMemcpy(arr_d,arr,size,hipMemcpyHostToDevice);

    while(n>1){
        max_per_block<<<no_of_blocks,THREADS_PER_BLOCK>>>(arr_d,b_d,n);
        n=(n+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK;
        hipMemcpy(arr_d,b_d,no_of_blocks*sizeof(int),hipMemcpyDeviceToDevice);
    }
    int ans;
    hipMemcpy(&ans,arr_d,sizeof(int),hipMemcpyDeviceToHost);
    cout<<ans;

    // clock_t cpu_start = clock();
    // clock_t cpu_stop = clock();
    // clock_t cpu_elapsed_time = 1000*(cpu_stop - cpu_start)/CLOCKS_PER_SEC;
    //int( pow(double(input[i]- *mean),2.0));
    // cudaEvent_t gpu_start,gpu_stop;

	// cudaEventCreate(&gpu_start);
	// cudaEventCreate(&gpu_stop);

	// cudaEventRecord(gpu_start,0);
    // cudaEventRecord(gpu_stop, 0);
	// cudaEventSynchronize(gpu_stop);
	// cudaEventElapsedTime(&gpu_elapsed_time, gpu_start, gpu_stop);
}