#include "hip/hip_runtime.h"
#include<iostream>
using namespace std;

#define THREADS_PER_BLOCK 256

__global__ void min_per_block(int *a_d,int *b_d,int n){
    int block = blockDim.x*blockIdx.x;
    int min=1000;
    for(int i=block;i<min(block+blockDim.x,n);i++){
        if(min>a_d[i]){
            min=a_d[i];
        }
    }
    b_d[blockIdx.x]=min;
}

int main() {
    int n;
    cout<<"Enter the no of elements";
    cin>>n;

    int *arr = new int[n];
    for(int i=0;i<n;i++){
        arr[i]=i+1;
    }
    
    int no_of_blocks = (n+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK;
    int size = n*sizeof(int);
    int *arr_d,*b_d;
    hipMalloc(&arr_d,size);
    hipMalloc(&b_d,no_of_blocks*sizeof(int));
    hipMemcpy(arr_d,arr,size,hipMemcpyHostToDevice);

    while(n>1){
        min_per_block<<<no_of_blocks,THREADS_PER_BLOCK>>>(arr_d,b_d,n);
        n=(n+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK;
        hipMemcpy(arr_d,b_d,no_of_blocks*sizeof(int),hipMemcpyDeviceToDevice);
    }
    int ans;
    hipMemcpy(&ans,arr_d,sizeof(int),hipMemcpyDeviceToHost);
    cout<<ans;
}