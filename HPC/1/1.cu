#include<iostream>
#include<cstdio>
#include<cstdlib>
#include<hip/hip_runtime.h>
using namespace std;


__global__ void minimum(int *input)
{
	int tid=threadIdx.x;
	auto step_size=1;
  int number_of_threads=blockDim.x;
  
  while(number_of_threads>0)
  {
      if(tid<number_of_threads)
      {
          int first=tid*step_size*2;
          int second=first+step_size;
          if(input[second]<input[first])
            input[first]=input[second];
      }
      step_size=step_size*2;
      number_of_threads/=2;
  }

}

__global__ void max(int *input)
{
   int tid=threadIdx.x;
   auto step_size=1;
   int number_of_threads=blockDim.x;
   
   while(number_of_threads>0)
   {
       if(tid<number_of_threads)
       {
           int first=tid*step_size*2;
           int second=first+step_size;
           if(input[second]>input[first])
            input[first]=input[second];
       }
       step_size*=2;
       number_of_threads/=2;
   }
}

__global__ void sum(int *input)
{
    const int tid=threadIdx.x;
    auto  step_size=1;
    int number_of_threads=blockDim.x;
    while(number_of_threads>0)
    {
        if(tid<number_of_threads)
        {
            const int first=tid*step_size*2;
            const int second=first+step_size;
            input[first]=input[first]+input[second];
        }
    step_size = step_size*2; 
		number_of_threads =number_of_threads/2;
    }
}

__global__ void average(int *input)
{
    const int tid=threadIdx.x;
    auto  step_size=1;
    int number_of_threads=blockDim.x;
    int totalElements=number_of_threads*2;
    while(number_of_threads>0)
    {
        if(tid<number_of_threads)
        {
            const int first=tid*step_size*2;
            const int second=first+step_size;
            input[first]=input[first]+input[second];
        }
        step_size = step_size*2;; 
		number_of_threads =number_of_threads/2;
    }
    input[0]=input[0]/totalElements;
}

int main()
{

	cout<<"Enter the no of elements"<<endl;
	int n;
	n=10;
  srand(n);
	int *arr=new int[n];
  int min=20000;
  
	for(int i=0;i<n;i++)
	{
		arr[i]=rand()%20000;
      if(arr[i]<min)
        min=arr[i];
    cout<<arr[i]<<" ";
	}

	int size=n*sizeof(int); 
	int *arr_d,result1;
	
  
  hipEvent_t start, stop;
  
	hipMalloc(&arr_d,size);
	hipMemcpy(arr_d,arr,size,hipMemcpyHostToDevice);

  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);
  
  minimum<<<1,n/2>>>(arr_d);
  hipMemcpy(&result1,arr_d,sizeof(int),hipMemcpyDeviceToHost);

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
    
  float elapsedTime;
  hipEventElapsedTime(&elapsedTime, start, stop);    
  elapsedTime /= 100.0f;
  
  cout<<"The minimum element is"<<result1<<endl;
  cout<<"average time elapsed:"<< elapsedTime;
  
  cout<<"The min element is"<<min;
   
    
  //#MAX OPERATION 
  int *arr_max,maxValue;
  hipMalloc(&arr_max,size);
	hipMemcpy(arr_max,arr,size,hipMemcpyHostToDevice);

  max<<<1,n/2>>>(arr_max);

	hipMemcpy(&maxValue,arr_max,sizeof(int),hipMemcpyDeviceToHost);

	cout<<"The maximum element is"<<maxValue<<endl;
    
  //#SUM OPERATION 
  int *arr_sum,sumValue;
  hipMalloc(&arr_sum,size);
	hipMemcpy(arr_sum,arr,size,hipMemcpyHostToDevice);

  sum<<<1,n/2>>>(arr_sum);

	hipMemcpy(&sumValue,arr_sum,sizeof(int),hipMemcpyDeviceToHost);

	cout<<"The sum of elements is"<<sumValue<<endl; 
   
  cout<<"The average of elements is"<<(sumValue/n)<<endl; 
  
 
   
  //#AVG OPERATION 
  int *arr_avg,avgValue;
  hipMalloc(&arr_avg,size);
	hipMemcpy(arr_avg,arr,size,hipMemcpyHostToDevice);

  average<<<1,n/2>>>(arr_avg);

	hipMemcpy(&avgValue,arr_avg,sizeof(int),hipMemcpyDeviceToHost);

	cout<<"The average of elements is "<<avgValue<<endl; 
  
   
 
  hipFree(arr_d);
  hipFree(arr_sum);
  hipFree(arr_max);
  hipFree(arr_avg);
    
    
 

return 0;

}