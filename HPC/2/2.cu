#include "hip/hip_runtime.h"
#include<iostream>
#include<time.h>
#include<cstdio>
#include<math.h>

#define N  2048
#define V  2048  //vector size
#define THREADS_PER_BLOCK 256
using namespace std;

__global__ void multiplication(int *a, int*b, int *c,int width){
   int row = blockIdx.y*width+threadIdx.y;
    int col = blockIdx.x*width+threadIdx.x;
    if(row<width && col <width) {
        float product_val = 0
        for(int k=0;k<width;k++) {
            product_val += d_M[row*width+k]*d_N[k*width+col];
        }
        d_p[row*width+col] = product_val;
    }
}


int main() {
    int *a, *b, *c;
    int *d_a, *d_b, *d_c;
    int Size = N * sizeof(int);
    int vectorSize = V *sizeof(int);

    a=(int *)malloc(vectorSize);
    b=(int *)malloc(size*vectorSize);
    c=(int *)malloc(vectorSize);

    srand(time(0));

	//initialize host vector by random elements
	for(int i=0;i<vectorSize;i++) {
		a[i] = rand();
    }
    
    //initialize matrix by random elements
	for(int i=0;i<N;i++) {
		for(int j=0;j<vectorSize;j++) {
            b[i*vectorSize+j] = rand();
            //In row-major layout, element(x,y) can be addressed as: x*width + y. 
            //Suppose the width of the matrix is 4. Then element (1,1) will be found at position −
            //1*4 + 1 = 5 in the 1D array.
		}
	}

    hipMalloc((void**)&d_a,vectorSize);
    hipMalloc((void**)&d_b, N*vectorSize);
    hipMalloc((void**)&d_c,vectorSize);

    hipMemcpy(d_a,a,vectorSize,hipMemcpyHostToDevice);
    hipMemcpy(d_b,b,N*vectorSize,hipMemcpyHostToDevice);

    multiplication<<<N,1>>>(d_a,d_b,d_c,N,vectorSize);

    hipDeviceSynchronize();

    hipMemcpy(c,d_c,size,hipMemcpyDeviceToHost);

    double error = 0;
	for(i = 0;i<N;i++) {
		double diff = double((a[i]+b[i])-c[i]);
		error+=diff;
    if(diff>0){
      printf("A+B=%d",a[i]+b[i]);
		  printf("C = %d",c[i]);
    }
       
	}

	error = sqrt(error);
    cout<<"error  = "<<error<<endl;
    
    free(a); free(b); free(c);
    hipFree(d_a);  hipFree(d_b); hipFree(d_c);

    return 0;
}