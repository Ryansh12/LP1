
#include <hip/hip_runtime.h>
#include<iostream>
#include<cstdio>
#include<math.h>

#define N (2048 * 2048)
#define THREADS_PER_BLOCK 256
using namespace std;

__global__ void add(int *a, int*b, int *c,int n){
    int index = threadIdx.x + blockIdx.x*blockDim.x;
    if(index<n){
        c[index] = a[index] + b[index];
    }
}


int main() {
    int *a, *b, *c;
    int *d_a, *d_b, *d_c;
    int size = N * sizeof(int);

    a=(int *)malloc(size);
    b=(int *)malloc(size);
    c=(int *)malloc(size);

    
	int i;
	for(i=0;i<N;i++) {
		a[i] = rand()%70;
		b[i] = rand()%70;
	}

    hipMalloc(&d_a,size);
    hipMalloc(&d_b,size);
    hipMalloc(&d_c,size);

    hipMemcpy(d_a,a,size,hipMemcpyHostToDevice);
    hipMemcpy(d_b,b,size,hipMemcpyHostToDevice);

    add<<<(N + THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(d_a,d_b,d_c,N);

    hipMemcpy(c,d_c,size,hipMemcpyDeviceToHost);

    double error = 0;
	for(i = 0;i<N;i++) {
		double diff = double((a[i]+b[i])-c[i]);
		error+=diff;
    if(diff>0){
      printf("A+B=%d",a[i]+b[i]);
		  printf("C = %d",c[i]);
    }
       
	}

	error = sqrt(error);
    cout<<"error  = "<<error<<endl;
    
    free(a); free(b); free(c);
    hipFree(d_a);  hipFree(d_b); hipFree(d_c);

    return 0;
}